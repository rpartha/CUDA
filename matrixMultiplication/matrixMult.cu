#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#define THREADS_PER_BLK 16 //aka block size

__global__ void mtimes(double *g_a, double *g_b, double *g_c, int m, int n, int k){ 
    int r = blockIdx.y * blockDim.y + threadIdx.y; 
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0; //thread-local variable
    if( c < k && r < m) {
        for(int i = 0; i < n; i++){
            sum += g_a[r * n + i] * g_b[i * k + c];
        }
        g_c[r * k + c] = sum; //write sum to global memory
    }                       
} 

/* square matrices using tiling*/
__global__ void mtimes_sq(double *gs_a, double *gs_b, double *gs_c, int m, int n, int k){
	__shared__ double s_a[THREADS_PER_BLK][THREADS_PER_BLK];
	__shared__ double s_b[THREADS_PER_BLK][THREADS_PER_BLK];

	unsigned int r = THREADS_PER_BLK * blockIdx.y + threadIdx.y;
    unsigned int c = THREADS_PER_BLK * blockIdx.x + threadIdx.x;
    
	unsigned int i, j;

	double x = 0.0;
	for (i = 0; i < (THREADS_PER_BLK + n - 1) / THREADS_PER_BLK; i++){
		if ((i * THREADS_PER_BLK + threadIdx.x < n) && (r < m))
		{
			s_a[threadIdx.y][threadIdx.x] = gs_a[(r * n) + (i * THREADS_PER_BLK) + threadIdx.x];
		} 
		else 
		{
			s_a[threadIdx.y][threadIdx.x] = 0.0;
		}

		if ((i * THREADS_PER_BLK + threadIdx.y < n) && (c < k)){
			s_b[threadIdx.y][threadIdx.x] = gs_b[c + k * (i * THREADS_PER_BLK + threadIdx.y)];
		}
		else{
			s_b[threadIdx.y][threadIdx.x] = 0.0;
		}
		__syncthreads();

		for (j = 0; j < THREADS_PER_BLK; j++){
			x += s_a[threadIdx.y][j] * s_b[j][threadIdx.x];
		}
		__syncthreads();
	}

	if ((r < m) && (c < k)){
		gs_c[(blockIdx.y * blockDim.y + threadIdx.y) * k + (blockIdx.x * blockDim.x) + threadIdx.x] = x;
    }
}

double *mtimes_gpu(double *a, double *b, int m, int n, int k){
    
    double *g_a, *g_b, *g_c;
    double *c = (double *) malloc(m * k * sizeof(double));

    hipMalloc((void**)&g_a, sizeof(double) * m * n); 
    hipMalloc((void**)&g_b, sizeof(double) * n * k); 
    hipMalloc((void**)&g_c, sizeof(double) * m * k); 

    hipMemcpy(g_a, a, m * n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(g_b, b, n * k * sizeof(double), hipMemcpyHostToDevice);

    dim3 dimBlock(THREADS_PER_BLK, THREADS_PER_BLK);
    dim3 dimGrid((int)ceil((double)m / (double)dimBlock.x), (int)ceil((double)k / (double)dimBlock.y));

    clock_t beg, end;
    beg = clock();

    mtimes_sq<<<dimGrid, dimBlock>>>(g_a, g_b, g_c, m, n, k);    

    hipDeviceSynchronize();

    end = clock();
    printf("Matrix multiplication in gpu (initial) took: %f secs\n", ((double)end - (double)beg) / CLOCKS_PER_SEC);

    hipMemcpy(c, g_c, sizeof(double) * m * k, hipMemcpyDeviceToHost);
	hipFree(g_a);
	hipFree(g_b);
	hipFree(g_c);

    return c;
}

double *mtimes_gpu_cublas(double *a, double *b, int m, int n, int k){

	double *g_rv, *g_cv, *g_c;

	double *temp = (double *) malloc(sizeof(double));
    double *c = (double *) malloc(m * k * sizeof(double));
    
	hipMalloc((void**)&g_rv, n * sizeof(double));
	hipMalloc((void**)&g_cv, n * sizeof(double));
	hipMalloc((void**)&g_c, sizeof(double));

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	int i, j, h;
	double *rv = (double *) malloc(n * sizeof(double));
	double *cv = (double *) malloc(n * sizeof(double));

	clock_t beg, end;
	double tot = 0;

	for (i = 0; i < m; i++){
		for (j = 0; j < k; j++){
			for (h = 0; h < n; h++){
				rv[h] = a[(i * n) + h];
				cv[h] = b[(h * k) + j];
			}
			hipblasSetVector(n, sizeof(double), rv, 1, g_rv, 1);
			hipblasSetVector(n, sizeof(double), cv, 1, g_cv, 1);
			hipblasGetVector(n, sizeof(double), g_rv, 1, rv, 1);
			hipblasGetVector(n, sizeof(double), g_cv, 1, cv, 1);

			beg = clock();

			hipblasDdot(handle, n, g_rv, 1, g_cv, 1, temp);
			hipDeviceSynchronize();

			end = clock();

			c[(i * k) + j] = *temp;
			
			tot += ((double)end - (double)beg);
		}
	}


	printf("Matrix Multiplication with CuBLAS Library took: %f seconds\n", tot / CLOCKS_PER_SEC);

	free(temp);
	hipFree(g_rv);
	hipFree(g_cv);
	hipblasDestroy(handle);

    return c; 
    
}

double *mtimes_gpu_cublas_func(double *a, double *b, int m, int n, int k){

    double *g_a, *g_b, *g_c;
    double *c = (double *) malloc(m * k * sizeof(double));

    hipMalloc((void**)&g_a, sizeof(double) * m * n); 
    hipMalloc((void**)&g_b, sizeof(double) * n * k); 
    hipMalloc((void**)&g_c, sizeof(double) * m * k); 

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipblasSetMatrix(m, n, sizeof(double), a, m, g_a, m);
	hipblasSetMatrix(n, k, sizeof(double), b, n, g_b, n);
    hipblasSetMatrix(m, k, sizeof(double), c, m, g_c, m);
    
    double alpha = 1.0;
    double beta = 1.0;

    clock_t beg, end;
    beg = clock();
    
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, k, n, &alpha, g_a, m, g_b, n, &beta, g_c, m);
    hipDeviceSynchronize();

    end = clock();
    printf("Matrix multiplication in gpu with CuBLAS gemm took: %f seconds\n", ((double)end - (double)beg) / CLOCKS_PER_SEC);

    hipblasGetMatrix(m, k, sizeof(double), g_c, m, c, m);

    hipFree(g_a);
	hipFree(g_b);
	hipFree(g_c);
	hipblasDestroy(handle);

    return c;
}

